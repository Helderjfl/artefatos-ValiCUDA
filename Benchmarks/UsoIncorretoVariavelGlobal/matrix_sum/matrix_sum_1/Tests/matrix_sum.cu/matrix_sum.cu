
#include <hip/hip_runtime.h>
#include <stdio.h>
__device__ void registerTrace(unsigned char *traceArray, int width, int id, int cont, unsigned char node)
{
	traceArray[cont * width + id] = node;
}

unsigned char history[2] = {0, 0};
__host__ void registerTraceHost(FILE *fp, unsigned char node)
{
	if (history[0] != node || history[1] != node)
		fprintf(fp, "%d-0\t", node);
	history[0] = history[1];
	history[1] = node;
}

__global__ void sumMatrix(int *d_a, int *d_b, int *d_c, int N, unsigned char * traceArray, int numberOfThreads) {
   int cont = 0;
   int map = (blockIdx.x  + gridDim.x  * blockIdx.y) * (blockDim.x * blockDim.y) + (threadIdx.x + blockDim.x * threadIdx.y);
   registerTrace(traceArray, numberOfThreads, map, cont++, 1);
   int idx = threadIdx.x + blockDim.x * blockIdx.x;
   int idy = threadIdx.y + blockDim.y * blockIdx.y;
   int pos = idx + idy * N;
   static __shared__ int s_a[32][32];
   static __shared__ int s_b[32][32];
   /* node 2 */
   if(idx < N && idy < N) {
      registerTrace(traceArray, numberOfThreads, map, cont++, 3);
      s_a[threadIdx.y][threadIdx.x] = d_a[pos];
      s_b[threadIdx.y][threadIdx.x] = d_b[pos];
   }
   registerTrace(traceArray, numberOfThreads, map, cont++, 4);
   /* node 5 */
   if(idx < N && idy < N) {
      registerTrace(traceArray, numberOfThreads, map, cont++, 6);
      if(s_a[threadIdx.y][threadIdx.x] < 0) {
         registerTrace(traceArray, numberOfThreads, map, cont++, 7);
         s_a[threadIdx.y][threadIdx.x] = s_a[threadIdx.y][threadIdx.x] * -1;
      }
      registerTrace(traceArray, numberOfThreads, map, cont++, 8);
      /* node 9 */
      if(s_b[threadIdx.y][threadIdx.x] < 0) {
         registerTrace(traceArray, numberOfThreads, map, cont++, 10);
         s_b[threadIdx.y][threadIdx.x] = s_b[threadIdx.y][threadIdx.x] * -1;
      }
      registerTrace(traceArray, numberOfThreads, map, cont++, 11);
      d_c[pos] = s_a[threadIdx.y][threadIdx.x] + s_b[threadIdx.y][threadIdx.x];
   }
   registerTrace(traceArray, numberOfThreads, map, cont++, 12);
}

int main(int argc, char *argv[]) {
   FILE *fp;
   FILE *fpHost = fopen("trace.Host.p0", "w");
   int N = atoi(argv[1]);
   registerTraceHost(fpHost, 1);
   size_t size = N * N * sizeof(int);
   int *h_a, *h_b, *h_c;
   h_a = (int *) malloc(size);
   h_b = (int *) malloc(size);
   h_c = (int *) malloc(size);
   int *d_a, *d_b, *d_c;
   /* node 2 */
   hipMalloc(&d_a, size);
   /* node 3 */
   hipMalloc(&d_b, size);
   /* node 4 */
   hipMalloc(&d_c, size);
   int i;
   for(/* node 5 */ i = 0; /* node 6 */ i < N * N; /* node 8 */ i++) {
      scanf("%d", &h_a[i]);
      registerTraceHost(fpHost, 7);
   }
   for(registerTraceHost(fpHost, 9), i = 0; /* node 10 */ i < N * N; /* node 12 */ i++) {
      scanf("%d", &h_b[i]);
      registerTraceHost(fpHost, 11);
   }
   registerTraceHost(fpHost, 13);
   hipMemcpy(d_a, h_a, size, hipMemcpyDeviceToHost); // bug
   /* node 14 */
   hipMemcpy(d_b, h_b, size, hipMemcpyHostToDevice);
   hipMemset(d_c, 0, size);
   dim3 gridsize(4, 4, 1);
   dim3 blocksize(32, 32, 1);
   int numberOfThreads0 = gridsize.x * gridsize.y * gridsize.z * blocksize.x * blocksize.y * blocksize.z;
   unsigned char *instTrace0;
   hipMallocManaged(&instTrace0, numberOfThreads0 * 2000 * sizeof(unsigned char));
   /* node 15 */
   sumMatrix<<<gridsize, blocksize>>>(d_a, d_b, d_c, N, instTrace0, numberOfThreads0);
   /* node 16 */
   hipMemcpy(h_c, d_c, size, hipMemcpyDeviceToHost);
   for(/* node 17 */ i = 0; /* node 18 */ i < N; registerTraceHost(fpHost, 24), i++) {
      int j;
      registerTraceHost(fpHost, 19);
      for(/* node 20 */ j = 0; /* node 21 */ j < N; /* node 23 */ j++) {
         printf("%d ", h_c[i * N + j]);
         registerTraceHost(fpHost, 22);
      }
      printf("\n");
   }
   registerTraceHost(fpHost, 25);
   hipFree(d_a);
   /* node 26 */
   hipFree(d_b);
   registerTraceHost(fpHost, 27);
   hipFree(d_c);
   free(h_a);
   free(h_b);
   free(h_c);
   fclose(fpHost);
   fp = fopen("trace.Grid.p1", "w");
   for(int threadIndex = 0; threadIndex < numberOfThreads0; threadIndex++)
   {
   	for(int nodeTrace = 0; nodeTrace < 2000; nodeTrace++)
   	{
   		if(instTrace0[threadIndex + nodeTrace * numberOfThreads0] != 0){
   			if(nodeTrace >= 2){
   				if(instTrace0[threadIndex + nodeTrace * numberOfThreads0] != instTrace0[threadIndex + (nodeTrace-1) * numberOfThreads0] ||
   				instTrace0[threadIndex + nodeTrace * numberOfThreads0] != instTrace0[threadIndex + (nodeTrace-2) * numberOfThreads0])
   					fprintf(fp, "%d-1\t", instTrace0[threadIndex + nodeTrace * numberOfThreads0]);
   			}else
   				fprintf(fp, "%d-1\t", instTrace0[threadIndex + nodeTrace * numberOfThreads0]);
   		}
   	}
   	fprintf(fp, "\n");
   }
   fclose(fp);
   fp = fopen("commsize", "w");
   fprintf(fp, "2");
   fclose(fp);
}

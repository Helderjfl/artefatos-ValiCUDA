
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void vectorAdd(int *d_a, int *d_b, int *d_c){
    int id = threadIdx.x;

    d_c[id] = d_a[id] + d_b[id];
}

int main(int argc, char **argv){
    int N = atoi(argv[1]);
    int i;
    int *a;
    int *b;
    int *c;

    size_t size = N*sizeof(int);

    a = (int*)malloc(size);
    b = (int*)malloc(size);
    c = (int*)malloc(size);
    
    for(i = 0; i < N; i++){
        a[i] = b[i] = i;
    }
    
    int *d_a;
    int *d_b;
    int *d_c;


    hipMalloc(&d_a, size);
    hipMalloc(&d_b, size);
    hipMalloc(&d_c, size);

    hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);

    vectorAdd<<<1, 1024>>>(d_a, d_b, d_c);
    hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);

    for(i = 0; i < N; i++)
    printf("%d ", c[i]);
    
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
}
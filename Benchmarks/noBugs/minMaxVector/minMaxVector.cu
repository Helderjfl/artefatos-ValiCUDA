/*
    Based on Mark Harris reduction algorithm.
    https://developer.download.nvidia.com/assets/cuda/files/reduction.pdf
*/

#include <hip/hip_runtime.h>
#include<stdio.h>
#include<math.h>

__global__ void findMax(int *g_data, int *g_odata, int N){
    __shared__ int sdata[1024];

    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    sdata[tid]=0;
    if(i < N)
        sdata[tid] = g_data[i];
    __syncthreads();

    int s;
    for(s = blockDim.x / 2; s > 0; s /= 2){
        if(tid < s){
            if(sdata[tid] < sdata[tid + s])
                sdata[tid] = sdata[tid + s];
        }
        __syncthreads();
    }

    if(tid == 0)
        g_odata[blockIdx.x] = sdata[0];
}

//Works for power of 2
__global__ void findMin(int *g_data, int *g_odata, int N){
    __shared__ int sdata[1024];

    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    sdata[tid]=0;
    if(i < N)
        sdata[tid] = g_data[i];
    __syncthreads();

    int s;
    for(s = blockDim.x / 2; s > 0; s /= 2){
        if(tid < s && i+s < N){ 
            if(sdata[tid] > sdata[tid + s])
                sdata[tid] = sdata[tid + s];
        }
        __syncthreads();
    }

    if(tid == 0)
        g_odata[blockIdx.x] = sdata[0];
}

int main(int argc, char* argv[]){
    int N = atoi(argv[1]);
    size_t size = N*sizeof(int);
    int limit;

    int *h_in, *h_out;
    h_in = (int*)malloc(size);
    h_out = (int*)malloc(size);

    int *g_data, *g_odata, *d_c;
    hipMalloc(&g_data, size);
    hipMalloc(&g_odata, size);
    hipMalloc(&d_c, size);

    srand(atoi(argv[1]));
    int i;
    for(i = 0; i < N; i++)
        scanf("%d", &h_in[i]);

    for(i = 0; i < N; i++)
        printf("%d ", h_in[i]);
    printf("\n");

    hipMemcpy(g_data, h_in, size, hipMemcpyHostToDevice);
    hipMemset(g_odata, 0, size);
    hipMemset(d_c, 0, size);

    int numBlock = 4;
    int numThread = 512;

    printf("Blocks: %d Threads: %d\n", numBlock, numThread);
    dim3 gridsize(numBlock,1,1);
    dim3 blocksize(numThread,1,1);

    limit = (int) ceil (((float) N) / blocksize.x);

    findMax<<<gridsize, blocksize>>>(g_data, g_odata, N);
    findMax<<<1, blocksize>>>(g_odata, d_c, limit);
    hipMemcpy(h_out, d_c, size, hipMemcpyDeviceToHost);
    printf("Max %d ", h_out[0]);

    findMin<<<gridsize, blocksize>>>(g_data, g_odata, N);
    findMin<<<1, blocksize>>>(g_odata, d_c, limit);
    hipMemcpy(h_out, d_c, size, hipMemcpyDeviceToHost);

    printf("Min %d \n", h_out[0]);
    return 0;
}
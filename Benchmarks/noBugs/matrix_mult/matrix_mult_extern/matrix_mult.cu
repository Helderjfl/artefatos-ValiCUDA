
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void multiMatrix (int *d_a, int *d_b, int *d_c, int N, int TILE_DIM) {
	int idx = threadIdx.x + blockDim.x * blockIdx.x;                
	int idy = threadIdx.y + blockDim.y * blockIdx.y;                
    int pos = idx + idy * N;                                        
	int temp_result = 0;  
	int posa, posb;                                          

	extern __shared__ int s[];
	int *s_a = s;
	int *s_b = &s[TILE_DIM * TILE_DIM];

	int tile_idx;
	for (tile_idx = 0; tile_idx < gridDim.x; tile_idx++) 
	{
        posa = idy * N + (tile_idx * TILE_DIM + threadIdx.x);
        posb = (tile_idx * TILE_DIM + threadIdx.y) * N + idx;
		
		if (posa < N*N) {
			s_a[threadIdx.y * TILE_DIM + threadIdx.x] = d_a[posa];
        } else {
            s_a[threadIdx.y * TILE_DIM + threadIdx.x] = 0;
        }
        if (posb < N*N) {
            s_b[threadIdx.y * TILE_DIM + threadIdx.x] = d_b[posb];
        } else {
            s_b[threadIdx.y * TILE_DIM + threadIdx.x] = 0;
        }
		__syncthreads();

		if (idx < N && idy < N) {
			int i;
			
			for (i = 0; i < TILE_DIM; i++) {
				if(s_a[threadIdx.y * TILE_DIM + i] != 0 && s_b[threadIdx.x + TILE_DIM * i] != 0){
					temp_result += s_a[threadIdx.y * TILE_DIM + i] * s_b[threadIdx.x + TILE_DIM * i]; 
				}
			}
		}
            
        __syncthreads();  
    }
    __syncthreads();   
    
    if(idx < N && idy < N)  {
		d_c[pos] = temp_result;
	} 
} 

int main (int argc, char* argv[]){
	int N = atoi(argv[1]); 
	int TILE_DIM = atoi(argv[2]);
	size_t size = N * N * sizeof(int);
	int num_thread, num_block;

	int *h_a, *h_b, *h_c;
	h_a = (int*)malloc(size);
	h_b = (int*)malloc(size);
	h_c = (int*)malloc(size);

	int *d_a, *d_b, *d_c;
	hipMalloc(&d_a, size);
	hipMalloc(&d_b, size);
	hipMalloc(&d_c, size);

    int i;
	for (i = 0; i < N*N; i++){
		scanf("%d", &h_a[i]);
	}

	for (i = 0; i < N*N; i++){
		scanf("%d", &h_b[i]);
	}
	
    hipMemcpy(d_a, h_a, size, hipMemcpyHostToDevice);
	hipMemcpy(d_b, h_b, size, hipMemcpyHostToDevice);
	hipMemset(d_c, 0, size);

	num_block = 4;
    num_thread = 16;

	dim3 gridsize(num_block, num_block, 1);
	dim3 blocksize(num_thread, num_thread, 1);
  
    multiMatrix<<<gridsize,blocksize, 2*TILE_DIM * TILE_DIM * sizeof(int)>>>(d_a, d_b, d_c, N, TILE_DIM);

    hipMemcpy(h_c, d_c, size, hipMemcpyDeviceToHost);

	for(i = 0; i < N; i++){
		int j;
		for(j = 0; j < N; j++)
			printf("%d ", h_c[i * N + j]);
		printf("\n");
	}

	hipFree(d_a);
	hipFree(d_b);
    hipFree(d_c);
    free(h_a);
    free(h_b);
    free(h_c);
}